#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "common.h"

__global__ void filterKernel(Pixel *input, Pixel *output, int width, int height)
{
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;

	int total = width * height;
	Pixel temp[filterarraysize-1];
	Pixel Biggest;
	Pixel tmp;
	int j = 0;

	if (y >= height || x >= width)
		return;

	for (int i = 0; i < filterarraysize; i++)// the pixels around (x,y) (how many depends on filtersize)
	{
		__syncthreads();
		int target_x = x -filtersize + (i % (filtersize*2+1));
		int target_y = y -filtersize + (i / (filtersize*2+1));
		if (target_x < 0 || target_x >= width || target_y < 0 || target_y >= height || i==4)
			continue;
		//Place the biggest pixel furthest in the array
		Biggest = input[target_x + target_y*width];
				for (int iter = 0; iter < j; iter++)
				{
					//If a bigger value then the current is found, switch place and continues search
					if (Biggest.value < temp[iter].value) {
						tmp = temp[iter];
						temp[iter] = Biggest;
						Biggest = tmp;
					}
				}
				temp[j] = Biggest;

		j++;

	}
	//Now that temp[] has all neighbors, take the pixel in the middle (the median)
	//If the amount of neighbors is even (n%2=0), take the avarge of the two middle pixels.
	if (j % 2 == 0)//Even
	{
		output[x+y*width].R = (temp[j / 2].R + temp[(j / 2 + 1)].R) / 2;
		output[x+y*width].G = (temp[j / 2].G + temp[(j / 2 + 1)].G) / 2;
		output[x+y*width].B = (temp[j / 2].B + temp[(j / 2 + 1)].B) / 2;
		output[x+y*width].value = (output[x+y*width].R + output[x+y*width].G + output[x+y*width].B) / 3;
	}
	else//Odd
		output[x+y*width] = temp[j / 2 + 1];
}


ImageData cudaFilter(ImageData imageInput)
{
	ImageData output;
	
	int height = output.height = imageInput.height;
	int width = output.width = imageInput.width;
	int total = height * width; 

	Pixel* matrix = new Pixel[height*width];	//Allocating a new matrix for the output
	
	Pixel *dev_input;
    Pixel *dev_matrix;

    // Allocate GPU buffers for three vectors (one input, one output).
    hipMalloc((void**)&dev_input, total * sizeof(Pixel));
    hipMalloc((void**)&dev_matrix, total * sizeof(Pixel));
    
    // Copy input vectors from host memory to GPU buffers.
	hipMemcpy(dev_input, imageInput.data, total * sizeof(Pixel), hipMemcpyHostToDevice);
    hipMemcpy(dev_matrix, matrix, total * sizeof(Pixel), hipMemcpyHostToDevice);
    
    // Launch a 2D-kernel on the GPU! 
	dim3 block(16,16);
	//dim3 grid ((total+15)/16, (total+15)/16);
	dim3 grid ((width+15)/16, (height+15)/16);
	filterKernel<<<grid,block>>>(dev_input, dev_matrix, width, height);
    
    // Copy output vector from GPU buffer to host memory.
    hipMemcpy(matrix, dev_matrix, total * sizeof(Pixel), hipMemcpyDeviceToHost);
    
	hipFree(dev_input);
	hipFree(dev_matrix);

	output.data = matrix;
	return output;   
}